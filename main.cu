#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "utility.h"
#include "cudrank.h"
#include "vec3.h"
#include "onb.h"
#include "ray.h"
#include "color.h"
#include "textureMat.h"
#include "hittable.h"
#include "material.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
// #include "ppm_to_png.h"
#include "aarect.h"
#include "box.h"
#include "path.h"
#include "stationary_path.h"
#include "straight_path.h"
#include "circular_path.h"
#include "moving_sphere.h"
#include "raw_img.h"


// #define STB_IMAGE_IMPLEMENTATION
// #include "stb_image.h"

#include "CImg.h"
using namespace cimg_library;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << hipGetErrorString(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}




__device__
color ray_color(ray &r,int depth,hittable_list **world,hiprandState* rand_state)
{	
	// color background(0.70, 0.80, 1.00);
	color background(0,0,0);
	ray cur_ray =r;
	color cur_attenuation =vec3(1.0,1.0,1.0);
	for(int i =0;i<depth;i++)
	{
		hit_record rec;
		if((*world)->hit(cur_ray,0.001,infinity,rec))
		{
			ray scattered;
			vec3 albedo;
			color emitted = rec.mat_ptr->emitted(rec.u,rec.v,rec.p);
			double pdf = 1;
			
			if (rec.mat_ptr->scatter(cur_ray, rec, albedo, scattered,pdf,rand_state))
			{
				cur_attenuation = albedo*rec.mat_ptr->scattering_pdf(r, rec, scattered)*cur_attenuation/pdf + emitted;
				cur_ray = scattered;
			}
			else
			{
				return cur_attenuation*emitted;
			}
		}
		else
		{
			return cur_attenuation * background;
		}
	}
	return vec3(0.0,0.0,0.0);
}

__global__
void process(vec3 *final_out,int image_width,int image_height,int sample_size,int max_depth,hittable_list** world,camera *cam)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	hiprandState thread_rand_state;
	hiprand_init(index,0,0,&thread_rand_state);
	for(int i=index;i<image_height*image_width;i+=stride)
	{

		int x = i%image_width;
		int y = (image_height-1) -(i/image_width);
		color pixel_color(0,0,0);
		for(int sample=0;sample<sample_size;sample++)
		{
			double u = double(x+random_double(&thread_rand_state)) / (image_width-1);
			double v = double(y+random_double(&thread_rand_state)) / (image_height-1);
	
			ray r = cam->get_ray(u,v);
			pixel_color += ray_color(r,max_depth,world,&thread_rand_state);
		}
		write_color(final_out,i,pixel_color,sample_size);
	}
}


__global__
void create_world(hittable_list **d_world,moving_sphere **move_list,raw_img** image_list,int frames,double running_time)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{

		hiprandState thread_rand_state ;
		hiprand_init(2020,0,0,&thread_rand_state);
		*d_world = new hittable_list(500);
	

		

		auto red   = new lambertian(new solid_color(color(.65, .05, .05)));
		auto blue   = new lambertian(new solid_color(color(3.0/255.0, 129.0/255.0, 231.0/255.0)));
		auto white = new lambertian(new solid_color(color(.73, .73, .73)));
		auto green = new lambertian(new solid_color(color(.12, .45, .15)));
		auto sunlight = new diffuse_light(new solid_color(25*color(1,1,1)));
		auto moonlight = new lambertian(new solid_color(color(0.5, 0.5, 0.5)));
		auto white_metal = new metal(color(1,1,1),0.2);





		auto mercury_img = new lambertian(new image_texture(image_list[0]->data,image_list[0]->width,image_list[0]->height));
		auto venus_img = new lambertian(new image_texture(image_list[1]->data,image_list[1]->width,image_list[1]->height));
		auto earth_img = new lambertian(new image_texture(image_list[2]->data,image_list[2]->width,image_list[2]->height));
		auto mars_img = new lambertian(new image_texture(image_list[3]->data,image_list[3]->width,image_list[3]->height));
		auto jupiter_img = new lambertian(new image_texture(image_list[4]->data,image_list[4]->width,image_list[4]->height));
		auto saturn_img = new lambertian(new image_texture(image_list[5]->data,image_list[5]->width,image_list[5]->height));


		auto test_sphere = new sphere(point3(0,-50,0),2,sunlight);

		auto mercury_orbit = new circular_path(point3(0,0,0),point3(1,0,0),point3(0,0,1),20,0.5,frames,random_double(&thread_rand_state,0,2*pi));
		auto venus_orbit = new circular_path(point3(0,0,0),point3(1,0,0),point3(0,0,1),30,0.2,frames,random_double(&thread_rand_state,0,2*pi));
		auto earth_orbit = new circular_path(point3(0,0,0),point3(1,0,0),point3(0,0,1),50,0.05,frames,random_double(&thread_rand_state,0,2*pi));
		auto mars_orbit = new circular_path(point3(0,0,0),point3(1,0,0),point3(0,0,1),70,0.03,frames,random_double(&thread_rand_state,0,2*pi));
		auto jupiter_orbit = new circular_path(point3(0,0,0),point3(1,0,0),point3(0,0,1),100,0.01,frames,random_double(&thread_rand_state,0,2*pi));
		auto saturn_orbit = new circular_path(point3(0,0,0),point3(1,0,0),point3(0,0,1),120,0.01,frames,1.7*pi);
		auto earth_moon_orbit =  new circular_path(earth_orbit,point3(1,2,0),point3(0,0,1),8,1,frames,random_double(&thread_rand_state,0,2*pi));
		

		auto sun = new sphere(point3(0,0,0),12,sunlight);
		auto mercury = new moving_sphere(mercury_orbit, 2, mercury_img);
		auto venus = new moving_sphere(venus_orbit, 3, venus_img);
		auto earth = new moving_sphere(earth_orbit, 4, earth_img);
		auto mars = new moving_sphere(mars_orbit, 3, mars_img);
		auto jupiter = new moving_sphere(jupiter_orbit, 9, jupiter_img);
		auto saturn = new moving_sphere(saturn_orbit, 7, saturn_img);
		auto earth_moon = new moving_sphere(earth_moon_orbit, 1, moonlight);


		// auto mirror = new xz_rect(-400, 400, -400,400, -50, white_metal);
		// auto earth = new moving_sphere(earth_orbit, 4, blue);
 		// (*d_world)->add(mirror);

		(*d_world)->add(mercury);
		(*d_world)->add(venus);
		(*d_world)->add(earth);
		(*d_world)->add(mars);
		(*d_world)->add(jupiter);
		(*d_world)->add(saturn);
		(*d_world)->add(earth_moon);


		(*d_world)->add(test_sphere);

		(*d_world)->add(sun);
		move_list[0] = mercury;
		move_list[1] = venus;
		move_list[2] = earth;
		move_list[3] = mars;
		move_list[4] = jupiter;
		move_list[5] = saturn;
		move_list[6] = earth_moon;
    }
}

__global__
void move_world(moving_sphere **move_list)
{
	if(threadIdx.x == 0 && blockIdx.x==0)
	{
		move_list[0]->move();
		move_list[1]->move();
		move_list[2]->move();
		move_list[3]->move();
		move_list[4]->move();
		move_list[5]->move();
		move_list[6]->move();
	}
	
}

// __global__
// void create_world(hittable_list **d_world,moving_sphere **move_list,raw_img** image_list,int frames,double running_time)
//  {
// 	if (threadIdx.x == 0 && blockIdx.x == 0)
// 	{

// 		hiprandState thread_rand_state ;
// 		hiprand_init(2020,0,0,&thread_rand_state);
// 		*d_world = new hittable_list(500);
		

// 		auto red   = new lambertian(new solid_color(color(.65, .05, .05)));
// 		auto white = new lambertian(new solid_color(color(.73, .73, .73)));
// 		auto green = new lambertian(new solid_color(color(.12, .45, .15)));
// 		auto light = new diffuse_light(new solid_color(color(15, 15, 15)));
// 		auto white_metal = new metal(color(1,0.7,1),0);


// 		(*d_world)->add(new yz_rect(0, 555, 0, 555, 555, green));
// 		(*d_world)->add(new yz_rect(0, 555, 0, 555, 0, red));
// 		(*d_world)->add(new xz_rect(213, 343, 227, 332, 554, light));
// 		(*d_world)->add(new xz_rect(0, 555, 0, 555, 0, white));
// 		(*d_world)->add(new xz_rect(0, 555, 0, 555, 555, white));
// 		(*d_world)->add(new xy_rect(0, 555, 0, 555, 555, white));	
// 		(*d_world)->add(new box(point3(130, 0, 65), point3(295, 165, 230), white));
// 		(*d_world)->add(new box(point3(265, 0, 295), point3(430, 330, 460), white));
//     }
// }


// __global__
// void move_world(moving_sphere **move_list)
// {
// 	if(threadIdx.x == 0 && blockIdx.x==0)
// 	{

// 	}
	
// }

__global__
void free_world(hittable_list **d_world)
{
	if(threadIdx.x == 0 && blockIdx.x==0)
	{
		delete (*d_world);
	}
}


int main()
{


	// Image
	const double aspect_ratio = 16.0/9.0;
	const int image_height = 720;
	const int image_width = static_cast<int>(image_height*aspect_ratio);
	const int sample_size = 2000;
	const int max_depth = 50;
	const int fps = 25;
	const double running_time = 10;
	const int frames = fps*running_time;





	vec3 *final_out = unified_ptr<vec3>(image_height*image_width*sizeof(vec3));

	// Camera
	point3 lookfrom(200,200,100);
	// point3 lookfrom(200,0,0);

    point3 lookat(0, 0, 0);
    vec3 vup(0,1,0);
	// point3 lookfrom(278, 278, -800);
    // point3 lookat(278, 278, 0);
    // vec3 vup(0,1,0);

	
	// camera *h_cam = new camera(lookfrom, lookat, vup, 40, aspect_ratio);
	// camera *d_cam = cuda_ptr<camera>(h_cam,sizeof(camera));
	// delete h_cam;
	
	camera *h_cam = new camera(lookfrom, lookat, vup, 40, aspect_ratio);
	camera *d_cam = cuda_ptr<camera>(h_cam,sizeof(camera));
	delete h_cam;
	


	//Kernel Parameters
	int block_size = 512;
	int num_blocks = ceil(double(image_width*image_height))/double(block_size);



	//World
	
	hittable_list **d_world;
	hipMalloc(&d_world, sizeof(hittable_list *));

	moving_sphere **move_list;
	hipMalloc(&move_list,10*sizeof(moving_sphere *));

	raw_img** image_list;
	hipMallocManaged(&image_list,10*sizeof(raw_img *));


	// Image Data
	



	CImg<unsigned char> mercury_img("data/mercurymap.jpg");
	image_list[0] = cuda_ptr<raw_img>(new raw_img(cuda_ptr<unsigned char>(mercury_img.data(),static_cast<size_t>(sizeof(unsigned char)*mercury_img.width()*mercury_img.height()*3)),mercury_img.width(),mercury_img.height()),sizeof(raw_img)) ;

	CImg<unsigned char> venus_img("data/venusmap.jpg");
	image_list[1] = cuda_ptr<raw_img>(new raw_img(cuda_ptr<unsigned char>(venus_img.data(),static_cast<size_t>(sizeof(unsigned char)*venus_img.width()*venus_img.height()*3)),venus_img.width(),venus_img.height()),sizeof(raw_img)) ;

	CImg<unsigned char> earth_img("data/earthmap.jpg");
	image_list[2] = cuda_ptr<raw_img>(new raw_img(cuda_ptr<unsigned char>(earth_img.data(),static_cast<size_t>(sizeof(unsigned char)*earth_img.width()*earth_img.height()*3)),earth_img.width(),earth_img.height()),sizeof(raw_img)) ;

	CImg<unsigned char> mars_img("data/marsmap.jpg");
	image_list[3] = cuda_ptr<raw_img>(new raw_img(cuda_ptr<unsigned char>(mars_img.data(),static_cast<size_t>(sizeof(unsigned char)*mars_img.width()*mars_img.height()*3)),mars_img.width(),mars_img.height()),sizeof(raw_img)) ;

	CImg<unsigned char> jupiter_img("data/jupitermap.jpg");
	image_list[4] = cuda_ptr<raw_img>(new raw_img(cuda_ptr<unsigned char>(jupiter_img.data(),static_cast<size_t>(sizeof(unsigned char)*jupiter_img.width()*jupiter_img.height()*3)),jupiter_img.width(),jupiter_img.height()),sizeof(raw_img)) ;

	CImg<unsigned char> saturn_img("data/saturnmap.jpg");
	image_list[5] = cuda_ptr<raw_img>(new raw_img(cuda_ptr<unsigned char>(saturn_img.data(),static_cast<size_t>(sizeof(unsigned char)*saturn_img.width()*saturn_img.height()*3)),saturn_img.width(),saturn_img.height()),sizeof(raw_img)) ;








	// if (!data) 
	// {
	// 	std::cerr << "ERROR: Could not load texture image file '" << filename << "'.\n";
	// 	width = height = 0;
	// }

    create_world<<<1,1>>>(d_world,move_list,image_list,fps,running_time);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	clock_t start,stop;
	start = clock();

	
	//Call Kernel
	
	char str[5];
	char* file_start = "outputppm/image";
	char* extension = ".ppm";
	char file_name[30];



	for(int j =0;j<frames;j++)
	{
		process<<<num_blocks,block_size>>>(final_out,image_width,image_height,sample_size,max_depth,d_world,d_cam);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		//File Handling and Importing ppm
		sprintf(str, "%d", j);
		strcpy(file_name,file_start);
		strcat(file_name,str);
		strcat(file_name,extension);

		FILE* file1 = fopen(file_name,"w");
	
		fprintf(file1,"P3 %d %d\n255\n",image_width,image_height);
	
		for (int i = 0; i<image_width*image_height; i++)
		{
			fprintf(file1,"%d %d %d\n",static_cast<int>(final_out[i][0]),static_cast<int>(final_out[i][1]),static_cast<int>(final_out[i][2]));
		}
	
		fclose(file1);
		move_world<<<1,1>>>(move_list);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		std::cerr<<(double(j+1)/double(frames))*100<<"%"<<"\n"<<std::flush;
	
	}




    stop = clock();
	double timer_seconds = ((static_cast<double>(stop - start))) / CLOCKS_PER_SEC;

	std::cerr<<"Done in "<<timer_seconds<<"s\n";
	//Free Memory

	free_world<<<1,1>>>(d_world);
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_cam));
	checkCudaErrors(hipFree(final_out));
	checkCudaErrors(hipDeviceReset());
	checkCudaErrors(hipDeviceSynchronize());


}
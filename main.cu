#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "utility.h"
#include "cudrank.h"
#include "vec3.h"
#include "ray.h"
#include "color.h"
#include "textureMat.h"
#include "hittable.h"
#include "material.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
// #include "ppm_to_png.h"
#include "aarect.h"
#include "box.h"
#include "path.h"
#include "straight_path.h"
#include "circular_path.h"
#include "moving_sphere.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << hipGetErrorString(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}




__device__
color ray_color(ray &r,int depth,hittable_list **world,hiprandState* rand_state)
{	
	// color background(0.70, 0.80, 1.00);
	color background(0,0,0);
	ray cur_ray =r;
	color cur_attenuation =vec3(1.0,1.0,1.0);
	for(int i =0;i<depth;i++)
	{
		hit_record rec;
		if((*world)->hit(cur_ray,0.001,infinity,rec))
		{
			ray scattered;
			vec3 attenuation;
			color emitted = rec.mat_ptr->emitted(rec.u,rec.v,rec.p);

			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered,rand_state))
			{
				cur_attenuation =attenuation*cur_attenuation + emitted;
				cur_ray = scattered;
			}
			else
			{
				return cur_attenuation*emitted;
			}
		}
		else
		{
			return cur_attenuation * background;
		}
	}
	return vec3(0.0,0.0,0.0);
}

__global__
void process(vec3 *final_out,int image_width,int image_height,int sample_size,int max_depth,hittable_list** world,camera *cam)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	hiprandState thread_rand_state;
	hiprand_init(index,0,0,&thread_rand_state);
	for(int i=index;i<image_height*image_width;i+=stride)
	{

		int x = i%image_width;
		int y = (image_height-1) -(i/image_width);
		color pixel_color(0,0,0);
		for(int sample=0;sample<sample_size;sample++)
		{
			double u = double(x+random_double(&thread_rand_state)) / (image_width-1);
			double v = double(y+random_double(&thread_rand_state)) / (image_height-1);
	
			ray r = cam->get_ray(u,v);
			pixel_color += ray_color(r,max_depth,world,&thread_rand_state);
		}
		write_color(final_out,i,pixel_color,sample_size);
	}
}

// __global__
// void create_world(hittable_list **d_world)
//  {
// 	if (threadIdx.x == 0 && blockIdx.x == 0)
// 	{

// 		hiprandState thread_rand_state ;
// 		hiprand_init(2020,0,0,&thread_rand_state);
// 		*d_world = new hittable_list(500);
		

// 		auto red   = new lambertian(new solid_color(color(.65, .05, .05)));
// 		auto white = new lambertian(new solid_color(color(.73, .73, .73)));
// 		auto green = new lambertian(new solid_color(color(.12, .45, .15)));
// 		auto light = new diffuse_light(new solid_color(color(15, 15, 15)));
// 		auto white_metal = new metal(color(1,0.7,1),0);


// 		(*d_world)->add(new yz_rect(0, 555, 0, 555, 555, white_metal));
// 		(*d_world)->add(new yz_rect(0, 555, 0, 555, 0, red));
// 		(*d_world)->add(new xz_rect(213, 343, 227, 332, 554, light));
// 		(*d_world)->add(new xz_rect(0, 555, 0, 555, 0, white));
// 		(*d_world)->add(new xz_rect(0, 555, 0, 555, 555, white));
// 		(*d_world)->add(new xy_rect(0, 555, 0, 555, 555, white));	
// 		(*d_world)->add(new box(point3(130, 0, 65), point3(295, 165, 230), white));
// 		(*d_world)->add(new box(point3(265, 0, 295), point3(430, 330, 460), white));
//     }
// }


__global__
void create_world(hittable_list **d_world,moving_sphere **move_list)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{

		hiprandState thread_rand_state ;
		hiprand_init(2020,0,0,&thread_rand_state);
		*d_world = new hittable_list(500);
		

		auto red   = new lambertian(new solid_color(color(.65, .05, .05)));
		auto white = new lambertian(new solid_color(color(.73, .73, .73)));
		auto green = new lambertian(new solid_color(color(.12, .45, .15)));
		auto light = new diffuse_light(new solid_color(color(15, 15, 15)));
		auto white_metal = new metal(color(1,0.7,1),0);
		auto sphere_path = new circular_path(point3(0,2,0),point3(3,2,0),point3(0,6,0),10,5,60,10);
		// auto sphere_path = new straight_path(point3(0,2,0),point3(3,2,0),0,2,5,1.0);

		auto sphere = new moving_sphere(sphere_path, 2, light);
		(*d_world)->add(sphere);
		move_list[0] = sphere;
		
    }
}

__global__
void move_world(moving_sphere **move_list)
{
	if(threadIdx.x == 0 && blockIdx.x==0)
	{
		move_list[0]->move();
	}
	
}

__global__
void free_world(hittable_list **d_world)
{
	if(threadIdx.x == 0 && blockIdx.x==0)
	{
		delete (*d_world);
	}
}


int main()
{


	// Image
	const double aspect_ratio = 16.0/9.0;
	const int image_height = 720;
	const int image_width = static_cast<int>(image_height*aspect_ratio);
	const int sample_size = 200;
	const int max_depth = 50;
	const int fps = 60;
	const double running_time = 10;
	const int frames = fps*running_time;




	vec3 *final_out = unified_ptr<vec3>(image_height*image_width*sizeof(vec3));

	// Camera
	point3 lookfrom(0,3,50);
    point3 lookat(0, 2, 0);
    vec3 vup(0,1,0);

	
	camera *h_cam = new camera(lookfrom, lookat, vup, 40, aspect_ratio);
	camera *d_cam = cuda_ptr<camera>(h_cam,sizeof(camera));
	delete h_cam;
	


	//Kernel Parameters
	int block_size = 512;
	int num_blocks = ceil(double(image_width*image_height))/double(block_size);



	//World
	
	hittable_list **d_world;
	hipMalloc(&d_world, sizeof(hittable_list *));

	moving_sphere **move_list;
	hipMalloc(&move_list,sizeof(moving_sphere *));


    create_world<<<1,1>>>(d_world,move_list);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	clock_t start,stop;
	start = clock();

	
	//Call Kernel
	
	char str[5];
	char* file_start = "outputppm/image";
	char* extension = ".ppm";
	char file_name[30];



	for(int j =0;j<frames;j++)
	{
		process<<<num_blocks,block_size>>>(final_out,image_width,image_height,sample_size,max_depth,d_world,d_cam);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		//File Handling and Importing ppm
		sprintf(str, "%d", j);
		strcpy(file_name,file_start);
		strcat(file_name,str);
		strcat(file_name,extension);

		FILE* file1 = fopen(file_name,"w");
	
		fprintf(file1,"P3 %d %d\n255\n",image_width,image_height);
	
		for (int i = 0; i<image_width*image_height; i++)
		{
			fprintf(file1,"%d %d %d\n",static_cast<int>(final_out[i][0]),static_cast<int>(final_out[i][1]),static_cast<int>(final_out[i][2]));
		}
	
		fclose(file1);
		move_world<<<1,1>>>(move_list);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		std::cerr<<j<<"/"<<frames<<"\n"<<std::flush;
	
	}




    stop = clock();
	double timer_seconds = ((static_cast<double>(stop - start))) / CLOCKS_PER_SEC;

	std::cerr<<"Done in "<<timer_seconds<<"s\n";
	//Free Memory

	free_world<<<1,1>>>(d_world);
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_cam));
	checkCudaErrors(hipFree(final_out));
	checkCudaErrors(hipDeviceReset());
	checkCudaErrors(hipDeviceSynchronize());


}